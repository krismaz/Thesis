#include "hip/hip_runtime.h"
/* 
 * File:   CUDA.cu
 * Author: kris
 *
 * Created on 03 December 2014, 17:25
 */

/**
@file CUDA.cu
 */
#include "Defines.h"
#include "CUDA.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <algorithm>
#include <hiprand/hiprand_kernel.h>
#include <ctime>

using namespace std;



//Following CUDA error checking code was taken from a StackOverflow thread
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }

inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true) {
    if (code != hipSuccess) {
        cerr << "GPUassert: " << hipGetErrorString(code) << file << line << " - " << code << endl;
        if (abort) exit(code);
    }
}
//End CUDA error checking from stackoverflow

__global__ void CudaHShakeUpDev(int * A, int offset, int max) {
    uint idx = blockIdx.y * blockDim.x * blockDim.y + blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= offset) return;
    for (int i = idx; i < max - offset; i += offset) {
        CE(A, i, i + offset);
    }
}

__global__ void CudaHShakeDownDev(int * A, int offset, int max) {
    uint idx = blockIdx.y * blockDim.x * blockDim.y + blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= offset) return;
    for (int i = max - offset + idx; i >= offset; i -= offset) {
        CE(A, i - offset, i);
    }
}

__global__ void CudaBitonicStep(int * A, int m, int ms, int NT) {
    uint idx = blockIdx.y * blockDim.x * blockDim.y + blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= NT) return;
    bool ascending = (idx & m - 1) < m >> 1; //idx % m < m / 2; 
    //int i = idx % (ms / 2) + ms * (idx / (ms / 2));
    int i = (idx & (ms - 1)) + ((idx << 1) & (~((ms << 1) - 1)));
    if (ascending) {
        CE(A, i, i + ms)
    } else {
        CE(A, i + ms, i)
    }
}

__global__ void CudaBitonicMultiStep(int * A, int NT) {
    uint bidx = blockIdx.y * blockDim.y + blockIdx.x;
    uint tidx = threadIdx.x;
    if (bidx * blockDim.x + tidx >= NT) return;
    int __shared__ cache[CUDAMULTISTEPSIZE];
    cache[tidx] = A[bidx * CUDAMULTISTEPSIZE + tidx];
    cache[tidx + CUDAMULTISTEPSIZE / 2] = A[bidx * CUDAMULTISTEPSIZE + tidx + CUDAMULTISTEPSIZE / 2];
    __syncthreads();
#pragma unroll
    for (int s = 1; s <= CUDAMULTISTEPSIZEB; s++) {
        uint sn = 1 << s;
        bool ascending = tidx % sn < sn / 2;
#pragma unroll //CUDA is not happy about this
        for (int m = s - 1; m >= 0; m--) {
            uint mn = 1 << m;
            uint i = (tidx & (mn - 1)) + ((tidx << 1) & (~((mn << 1) - 1)));
            if (ascending) {
                CE(cache, i, i + mn)
            } else {
                CE(cache, i + mn, i)
            }
            __syncthreads();
        }
    }
    A[bidx * CUDAMULTISTEPSIZE + tidx] = cache[tidx];
    A[bidx * CUDAMULTISTEPSIZE + tidx + CUDAMULTISTEPSIZE / 2] = cache[tidx + CUDAMULTISTEPSIZE / 2];
}

__global__ void CudaBitonicMultiMergeStep(int * A, uint sn, int NT) {
    uint bidx = blockIdx.y * blockDim.y + blockIdx.x;
    uint tidx = threadIdx.x;
    if (bidx * blockDim.x + tidx >= NT) return;
    int __shared__ cache[CUDAMULTISTEPSIZE];
    cache[tidx] = A[bidx * CUDAMULTISTEPSIZE + tidx];
    cache[tidx + CUDAMULTISTEPSIZE / 2] = A[bidx * CUDAMULTISTEPSIZE + tidx + CUDAMULTISTEPSIZE / 2];
    __syncthreads();
    uint idx = blockIdx.y * blockDim.x * blockDim.y + blockIdx.x * blockDim.x + threadIdx.x;
    bool ascending = idx % sn < sn / 2;
#pragma unroll //CUDA is not happy about this
    for (int m = CUDAMULTISTEPSIZEB; m >= 0; m--) {
        uint mn = 1 << m;
        uint i = (tidx & (mn - 1)) + ((tidx << 1) & (~((mn << 1) - 1)));
        if (ascending) {
            CE(cache, i, i + mn)
        } else {
            CE(cache, i + mn, i)
        }
        __syncthreads();
    }
    A[bidx * CUDAMULTISTEPSIZE + tidx] = cache[tidx];
    A[bidx * CUDAMULTISTEPSIZE + tidx + CUDAMULTISTEPSIZE / 2] = cache[tidx + CUDAMULTISTEPSIZE / 2];
}

__global__ void CudaShuffleStep(int * A, hiprandState *state, int seed, int ms, int NT) { //No shuffling done here
    uint idx = blockIdx.y * blockDim.x * blockDim.y + blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= NT) return;
    if (ms == NT * 2) {
        if (idx < 512) {
            hiprand_init(seed, idx, 0, &state[idx]);
        }
        return;
    }
    int r = hiprand(&(state[idx % 512]));
    int i = idx % (ms / 2) + ms * (idx / (ms / 2));
    //int i = (idx & (ms-1)) + ((idx << 1) & (~((ms<<1)-1)));
    if (r % 2 == 1) {
        int temp = A[i];
        A[i] = A[i + ms];
        A[i + ms] = temp;
    }
}

__global__ void CudaMergeStep(int * A, int p, int r, int d, int NT) {
    uint idx = blockIdx.y * blockDim.x * blockDim.y + blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= NT) return;
    if ((idx & p) == r) {
        CE(A, idx, idx + d);
    }
}

__global__ void CudaOddEvenMultiMergeStep(int * A, int NT) {
    uint bidx = blockIdx.y * blockDim.y + blockIdx.x;
    uint tidx = threadIdx.x;
    if (bidx * blockDim.x + tidx >= NT) return;
    int __shared__ cache[CUDAMULTISTEPSIZE/2];
    cache[tidx] = A[bidx * CUDAMULTISTEPSIZE/2 + tidx];
    __syncthreads();
    for (int p = CUDAMULTISTEPSIZE / 4; p > 0; p /= 2) {
        int d = p;
        int r = 0;
        for (int q = CUDAMULTISTEPSIZE / 4; q >= p; q /= 2) {
            int H = CUDAMULTISTEPSIZE/2  - d;

            
            if ((tidx & p) == r && tidx < H) {
                CE(cache, tidx, tidx + d);
            }

            d = q - p;
            r = p;
            __syncthreads();
        }
    }
    __syncthreads();
    A[bidx * CUDAMULTISTEPSIZE/2 + tidx] = cache[tidx];
}

texture<int, 1, hipReadModeElementType> perm;

__global__ void CudaRegionCompareDev(int* A, int r2, int jump) {
    uint idx = blockIdx.y * blockDim.x * blockDim.y + blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= jump) return;
    int id = tex1Dfetch(perm, idx);
    CE(A, id, r2 * jump + idx);
}

__host__ int* ToDevice(int * A, int N) {
    int* d_A;
    hipMalloc(&d_A, N * sizeof (int));
    hipMemcpy(d_A, A, N * sizeof (int), hipMemcpyHostToDevice);
    return d_A;
}

__host__ void FromDevice(int * dev, int* host, int N) {
    hipMemcpy(host, dev, N * sizeof (int), hipMemcpyDeviceToHost);
    hipFree(dev);
}

__host__ int * OnDevice(int N) {
    int* d_A;
    hipMalloc(&d_A, N * sizeof (int));
    return d_A;
}

__host__ void DestroyDevice(int* dev) {
    hipFree(dev);
}

__host__ void MoveToDevice(int* dev, int * host, int N) {
    hipMemcpy(dev, host, N * sizeof (int), hipMemcpyHostToDevice);
}

__host__ void CudaHShakeUp(int* A, int H, int N) {
    dim3 grid_size;
    grid_size.x = min(CUDABLOCKX, H / CUDATHREADS + (H % CUDATHREADS != 0));
    grid_size.y = H / (CUDABLOCKX * CUDATHREADS) + (H % (CUDABLOCKX * CUDATHREADS) != 0);

    CudaHShakeUpDev << <grid_size, min(CUDATHREADS, H)>>>(A, H, N);
    //gpuErrchk( hipPeekAtLastError() );
    //gpuErrchk( hipDeviceSynchronize() );
}

__host__ void CudaHShakeDown(int* A, int H, int N) {
    dim3 grid_size;
    grid_size.x = min(CUDABLOCKX, H / CUDATHREADS + (H % CUDATHREADS != 0));
    grid_size.y = H / (CUDABLOCKX * CUDATHREADS) + (H % (CUDABLOCKX * CUDATHREADS) != 0);

    CudaHShakeDownDev << <grid_size, min(CUDATHREADS, H)>>>(A, H, N);
    //gpuErrchk( hipPeekAtLastError() );
    //gpuErrchk( hipDeviceSynchronize() );
}

__host__ void CudaBitonicSort(int* A, int N) {
    int H = N / 2;
    dim3 grid_size;
    grid_size.x = min(CUDABLOCKX, H / CUDATHREADS + (H % CUDATHREADS != 0));
    grid_size.y = H / (CUDABLOCKX * CUDATHREADS) + (H % (CUDABLOCKX * CUDATHREADS) != 0);
    dim3 sgrid_size;
    sgrid_size.x = min(CUDABLOCKX, H / (CUDAMULTISTEPSIZE / 2) + (H % (CUDAMULTISTEPSIZE / 2) != 0));
    sgrid_size.y = H / (CUDABLOCKX * (CUDAMULTISTEPSIZE / 2)) + (H % (CUDABLOCKX * (CUDAMULTISTEPSIZE / 2)) != 0);

    CudaBitonicMultiStep << <sgrid_size, CUDAMULTISTEPSIZE / 2 >> >(A, H);
    for (int i = CUDAMULTISTEPSIZE; i <= N; i *= 2) { //Sorting size

        for (int ms = i / 2; ms > 0; ms /= 2) {
            if (ms == CUDAMULTISTEPSIZE / 2) {
                CudaBitonicMultiMergeStep << <sgrid_size, CUDAMULTISTEPSIZE / 2 >> >(A, i, H);
                break;
            }
            CudaBitonicStep << <grid_size, CUDATHREADS>>>(A, i, ms, H);
        }
    }
    gpuErrchk(hipPeekAtLastError());
    gpuErrchk(hipDeviceSynchronize());
}

__host__ void CudaOddEvenMergeSort(int* A, int N) {
    //Based Chinese scientists

    for (int p = N / 2; p > 0; p /= 2) {
        int d = p;
        int r = 0;
        for (int q = N / 2; q >= p; q /= 2) {
            int H = N - d;
            dim3 grid_size;
            grid_size.x = min(CUDABLOCKX, H / CUDATHREADS + (H % CUDATHREADS != 0));
            grid_size.y = H / (CUDABLOCKX * CUDATHREADS) + (H % (CUDABLOCKX * CUDATHREADS) != 0);

            CudaMergeStep << <grid_size, CUDATHREADS>>>(A, p, r, d, H);

            d = q - p;
            r = p;
        }
    }
    gpuErrchk(hipPeekAtLastError());
    gpuErrchk(hipDeviceSynchronize());
    
}

__host__ void CudaShuffle(int* A, int N) { //This doesn't shuffle
    int H = N / 2;
    dim3 grid_size;
    grid_size.x = min(CUDABLOCKX, H / CUDATHREADS + (H % CUDATHREADS != 0));
    grid_size.y = H / (CUDABLOCKX * CUDATHREADS) + (H % (CUDABLOCKX * CUDATHREADS) != 0);
    hiprandState *devStates;
    hipMalloc(&devStates, 512 * sizeof (hiprandState));
    for (int i = N; i > 0; i /= 2) {
        CudaShuffleStep << <grid_size, CUDATHREADS>>>(A, devStates, time(0), i, H);
    }
    gpuErrchk(hipPeekAtLastError());
    gpuErrchk(hipDeviceSynchronize());

}

__host__ void CudaRegionCompare(int* A, int r2, int * d_perm, int jump) {
    hipBindTexture(0, perm, d_perm, jump * sizeof (int));
    dim3 grid_size;
    grid_size.x = min(CUDABLOCKX, jump / CUDATHREADS + (jump % CUDATHREADS != 0));
    grid_size.y = jump / (CUDABLOCKX * CUDATHREADS) + (jump % (CUDABLOCKX * CUDATHREADS) != 0);
    CudaRegionCompareDev << <grid_size, CUDATHREADS>>>(A, r2, jump);
    //gpuErrchk(hipPeekAtLastError());
    //gpuErrchk(hipDeviceSynchronize());
    hipUnbindTexture(perm);
}